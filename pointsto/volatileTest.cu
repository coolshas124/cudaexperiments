#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>


#include <hiprand/hiprand_kernel.h>

#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>

using namespace std;

__device__ int sum = 1;

__global__ void degreeCalc (int *array){
	
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i>=1000000){
		return;
	}
	
	sum+=array[i];
	
//	if (i==999999){
//		printf("%d", sum);
//	}
}


int main(int argc, char const *argv[])
{
	/* code */
	
	int n = 1000000;
	
	int *h_array = new int [n];
	
	int *h_sum = new int;
	
	int *d_array = NULL;
    hipMalloc((void **)&d_array, n*sizeof(int));
    	
    	
	for (int i = 0; i < n; ++i)
	{
		/* code */
		h_array[i]=1;
	}

	hipMemcpy(d_array, h_array, n*sizeof(int), hipMemcpyHostToDevice);
	
	int threadsPerBlock = 512;
	int blocksPerGrid = (n + threadsPerBlock -1)/threadsPerBlock;
	
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
 
	hipEventRecord(start, 0);
 		
	degreeCalc<<<blocksPerGrid, threadsPerBlock>>>(d_array);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	hipMemcpyFromSymbol(h_sum, HIP_SYMBOL(sum), sizeof(int), 0, hipMemcpyDeviceToHost);

	cout<<*h_sum<<endl;

	// Retrieve result from device and store it in host array
	hipEventElapsedTime(&time, start, stop);
	cout<<"Time for the kernel: "<<time<<" ms"<<endl;

	
	delete[] h_array;
	hipFree(d_array);
	
	hipDeviceReset();
	return 0;
}
