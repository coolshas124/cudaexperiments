#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <fstream>

#include <hiprand/hiprand_kernel.h>

#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>

#define bucketLimitDecr 600
#define bucketLimitIncr 1400

using namespace std;


__device__ int d_count = 0;
__device__ int d_countNew = 0;

__global__ void colourCountFunc (int *colouring, int n, int *propagationArray){
	int i= blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i>=n){
		return;
	}
	
	propagationArray[colouring[i]-1]=1;
}

__global__ void propagationColouringNewest (int *vertexArray, int *neighbourArray, int *numbers, int n, int m, int *colouring, int *propagationArray){

	int i= blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i>=n){
		return;
	}
	
	if (propagationArray[i]!=0){
		return;
	}
	
	int myMax = numbers[i];

//	printf("I am node %d with value %d\n", i+1, myMax);
	
	int start = -1, stop = -1;
	
	start = vertexArray[i];
	
	
	stop = vertexArray[i+1];
	
	for (int j=start; j<stop; j++){
		
//		printf("My neighbour %d with value %d from %d \n", neighbourArray[j], numbers[neighbourArray[j]-1], i+1);
	
		int neighbour = neighbourArray[j]-1;
		
		if (propagationArray[neighbour]==0 && numbers[neighbour] >= myMax){
			if (numbers[neighbour] == myMax){
				if (i < neighbour){
					continue;
				}
			}
			
			return;
		}
	}
	
	propagationArray[i]=1;
	atomicAdd(&d_countNew, 1);
	
	int colours=0;
	
	bool bucket[bucketLimitDecr];
	
	int colouringLimit = colouring[i];
	
	for (int j=0; j<colouringLimit-1; j++){
		bucket[j]=true;
	}	
	
	for (int j=start; j<stop; j++){
		if (neighbourArray[j]==0){
			continue;
		}
		
		int bucketIndex = colouring[neighbourArray[j]-1]; 
		
		if (bucketIndex < colouringLimit){
			bucket[bucketIndex-1] = false;
		}
		
	}
	
	for (int j=0; j<colouringLimit-1; j++){
		if(bucket[j]){
			colours=j+1;
			break;
		}
	}
	
	
	if (colours >= colouringLimit){
		printf("R DANGER DANGER DANGER DANGER DANGER DANGER DANGER\n");
	}
	
	
	if (!colours){
		return;
	}
	
	colouring[i]=colours;

}

__global__ void colourMinMax (int *vertexArray, int *neighbourArray, int *numbers, int n, int m, int *colouring, int currentColour){

	int i= blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i>=n){
		return;
	}
	
	if (colouring[i]!=0){
		return;
	}
	
	int myValue = numbers[i];

//	printf("I am node %d with value %d\n", i+1, myMax);
	
	int start = -1, stop = -1;
	
	start = vertexArray[i];
	
	
	stop = vertexArray[i+1];
	
	
	bool max = true, min = true;
	
	for (int j=start; j<stop; j++){
		
//		printf("My neighbour %d with value %d from %d \n", neighbourArray[j], numbers[neighbourArray[j]-1], i+1);
	
		int neighbour = neighbourArray[j]-1;
		
		if (max && colouring[neighbour]==0 && numbers[neighbour] >= myValue){
			if (numbers[neighbour] == myValue){
				if (i < neighbour){
					continue;
				}
			}
			
			max=false;
			
			if (!min){
				return;
			}
		}
		
		if (min && colouring[neighbour]==0 && numbers[neighbour] <= myValue){
			if (numbers[neighbour] == myValue){
				if (i > neighbour){
					continue;
				}
			}
			
			min=false;
			
			if (!max){
				return;
			}
		}
	}
	
	if (max){
		colouring[i] = currentColour;	
	}
	else if (min){
		colouring[i] = currentColour+1;
	}
	
	atomicAdd(&d_count, 1);
}

__global__ void setup_kernel (hiprandState * state, unsigned long seed ){

    int i= blockDim.x * blockIdx.x + threadIdx.x;

    hiprand_init (seed, i, 0, &state[i]);
} 

__global__ void randomNumbering (hiprandState* globalState, int *degreeCount, int n, int limit){

	int i= blockDim.x * blockIdx.x + threadIdx.x;
	
	hiprandState localState = globalState[i];
    float RANDOM = hiprand_uniform( &localState );
    globalState[i] = localState;
    
    RANDOM *= (limit - 1 + 0.999999);
    RANDOM += 1;
	
	degreeCount[i] = (int) RANDOM;
}


__global__ void degreeCalc (int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m){

	int i= blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i>=n){
		return;
	}
	
	
	int start = -1, stop = -1;
	int diff=0;
	
	start = vertexArray[i];
	
	
	stop = vertexArray[i+1];
	

	diff = stop-start;
		
	degreeCount[i]=diff;
}

void edgesPrint (int vertexArray[], int neighbourArray[], int n, int m){ 

	for (int i=0; i<n-1; i++){
		for (int j = vertexArray[i]; j < vertexArray[i+1]; ++j){

				cout<<"e "<<i+1<<" "<<neighbourArray[j]<<endl;
				/* code */
		}
	}

	for (int j = vertexArray[n-1]; j < m; ++j)
	{
			cout<<"e "<<n<<" "<<neighbourArray[j]<<endl;
				/* code */
		
	}
}

int main(int argc, char const *argv[])
{
	string a, b;
	int n, m;
	
	cin>>n>>m;
	
	ofstream fout;
	fout.open("output7.txt",ios::app);
	
	int *h_count = new int;

	int *h_vertexArray = new int [n+1];
	int *h_neighbourArray = new int [2*m];
	int *h_degreeCount = new int [n];
	int *h_colour = new int [n];
	
	int *h_propagationArray1 = new int [n];
	int *h_propagationArray2 = new int [n];
	
	int *d_propagationArray1 = NULL;
    hipMalloc((void **)&d_propagationArray1, (1400)*sizeof(int));
    hipMemset((void *)d_propagationArray1, 0, (1400)*sizeof(int));
    	
    int *d_propagationArray2 = NULL;
    hipMalloc((void **)&d_propagationArray2, (n)*sizeof(int));
    hipMemset((void *)d_propagationArray2, 0, (n)*sizeof(int));
	
	int *d_vertexArray = NULL;
    hipMalloc((void **)&d_vertexArray, (n+1)*sizeof(int));
    	
    int *d_neighbourArray = NULL;
    hipMalloc((void **)&d_neighbourArray, 2*m*sizeof(int));
    	
    int *d_colour = NULL;
    hipMalloc((void **)&d_colour, (n)*sizeof(int));
    hipMemset((void *)d_colour, 0, (n)*sizeof(int));
    	
    int *d_degreeCount = NULL;
    hipMalloc((void **)&d_degreeCount, (n)*sizeof(int));
    hipMemset((void *)d_degreeCount, 0, (n)*sizeof(int));
    	
    hiprandState* devStates;
    hipMalloc ( &devStates, n*sizeof( hiprandState ) );
    	
	for (int i = 0; i < n+1; ++i)
	{
		h_vertexArray[i]=2*m;
	}

	int NSlast = 0;
	int NSoffset = 0;
	int NSprev=0;
	
	
	for (int i=0; i<2*m; i++){
		int start, end;
		cin>>start>>end;
		
		for (int j=NSlast+1; j<start; j++){
			h_vertexArray[j-1]=NSoffset;
			
		}
		
		if (NSprev!=start){
			NSlast=start;
			h_vertexArray[start-1]=NSoffset;
			NSprev=start;
		}
		
		h_neighbourArray[NSoffset]=end;
		NSoffset++;
		
	}

	hipEvent_t start, stop;
	float timeNew;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);
	
	hipMemcpy(d_vertexArray, h_vertexArray, (n+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_neighbourArray, h_neighbourArray, 2*m*sizeof(int), hipMemcpyHostToDevice);
	
	int threadsPerBlock = 512;
	int blocksPerGrid = (n + threadsPerBlock -1)/threadsPerBlock;
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&timeNew, start, stop);
	fout<<timeNew<<"\t";
	
	degreeCalc<<<blocksPerGrid, threadsPerBlock>>>(d_vertexArray, d_neighbourArray, d_degreeCount, n, m);
	
	thrust::device_ptr<int> d_ptr = thrust::device_pointer_cast(d_degreeCount);
  	int maxDegree = *(thrust::max_element(d_ptr, d_ptr + n));

	cout<<"Max = "<<maxDegree<<endl;
	
	hipEventRecord(start, 0);
	
	setup_kernel <<<blocksPerGrid, threadsPerBlock>>> ( devStates, time(NULL) );
	
	randomNumbering<<<blocksPerGrid, threadsPerBlock>>>(devStates, d_degreeCount, n, n);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&timeNew, start, stop);
	fout<<timeNew<<"\t";

//	hipMemcpy(h_degreeCount, d_degreeCount, n*sizeof(int), hipMemcpyDeviceToHost);

//	cout<<"Random numbers: "<<endl;
//	
//	for (int i=0; i<n; i++){
//		cout<<h_degreeCount[i]<<endl;
//	}

	int colourCount = 1;
	
	hipEventRecord(start, 0);
	
	while (1){
		colourMinMax<<<blocksPerGrid, threadsPerBlock>>>(d_vertexArray, d_neighbourArray, d_degreeCount, n, m, d_colour, colourCount);
	
		hipMemcpyFromSymbol(h_count, HIP_SYMBOL(d_count), sizeof(int), 0, hipMemcpyDeviceToHost);
		
		//cout<<"H Count = "<<*h_count<<"at colour: "<<colourCount<<endl;
		
		if (*h_count == n){
			break;
		}
		
		colourCount+=2;
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&timeNew, start, stop);
	fout<<timeNew<<"\t"<<(colourCount+1)/2<<"\t";
	
	thrust::device_ptr<int> c_ptr = thrust::device_pointer_cast(d_colour);
  	int maxColour = *(thrust::max_element(c_ptr, c_ptr + n));

	cout<<"Max Colour = "<<maxColour<<endl;
  	
  	fout<<maxColour<<"\t";
  	
  	int maxColourNew;
	thrust::device_ptr<int> d_propagationArray_ptr = thrust::device_pointer_cast(d_propagationArray1);
	
	
	maxColourNew = 0;
	
	colourCountFunc<<< blocksPerGrid, threadsPerBlock >>>(d_colour, n, d_propagationArray1);
	
	maxColourNew = thrust::reduce(d_propagationArray_ptr, d_propagationArray_ptr + 1400);
	
	hipMemset((void *)d_propagationArray1, 0, (1400)*sizeof(int));
	
	fout<<maxColourNew<<"\t";
	
	hipEventRecord(start, 0);
	
	while (1){
		propagationColouringNewest<<<blocksPerGrid, threadsPerBlock>>>(d_vertexArray, d_neighbourArray, d_degreeCount, n, m, d_colour, d_propagationArray2);
	
		hipMemcpyFromSymbol(h_count, HIP_SYMBOL(d_countNew), sizeof(int), 0, hipMemcpyDeviceToHost);
		
//		cout<<"H Count = "<<*h_count<<endl;
		
		if (*h_count == n){
			break;
		}
		
	}
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&timeNew, start, stop);
	
	
	
	fout<<timeNew<<"\t";
	
	maxColour = *(thrust::max_element(c_ptr, c_ptr + n));

	cout<<"Max Colour = "<<maxColour<<endl;
	
	fout<<maxColour<<"\t";
	
	maxColourNew = 0;
	
	colourCountFunc<<< blocksPerGrid, threadsPerBlock >>>(d_colour, n, d_propagationArray1);
	
	maxColourNew = thrust::reduce(d_propagationArray_ptr, d_propagationArray_ptr + 1400);
	
	hipMemset((void *)d_propagationArray1, 0, (1400)*sizeof(int));
	
	fout<<maxColourNew<<"\t";
	
	hipEventRecord(start, 0);
	
  	hipMemcpy(h_colour, d_colour, n*sizeof(int), hipMemcpyDeviceToHost);
  	
  	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&timeNew, start, stop);
	fout<<timeNew<<"\n";
  	
//  	cout<<"Colour numbers: "<<endl;
//	
//	for (int i=0; i<n; i++){
//		cout<<h_colour[i]<<endl;
//	}

	delete h_count;		
	delete[] h_vertexArray;
	delete[] h_neighbourArray;
	delete[] h_degreeCount;
	delete[] h_colour;
	
	hipFree(d_neighbourArray);
	hipFree(d_vertexArray);
	hipFree(d_degreeCount);
	hipFree(d_colour);
	
	hipDeviceReset();
	return 0;
}
