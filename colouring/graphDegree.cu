#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>

#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>

using namespace std;

__global__ void degreeCalc (int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m){

	int i= blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i>=n){
		return;
	}
	
	
	int start = -1, stop = -1;
	int diff=0;
	
	start = vertexArray[i];
	
	if (i==n-1){	
		stop = m;
	}
	
	else{
		stop = vertexArray[i+1];
	}

	diff = stop-start;
		
	atomicAdd(&degreeCount[i], diff);
	
	for (int j=start; j<stop; j++){
		atomicAdd(&degreeCount[neighbourArray[j]-1], 1);
	}

}

void edgesPrint (int vertexArray[], int neighbourArray[], int n, int m){ 

	for (int i=0; i<n-1; i++){
		for (int j = vertexArray[i]; j < vertexArray[i+1]; ++j){

				cout<<"e "<<i+1<<" "<<neighbourArray[j]<<endl;
				/* code */
		}
	}

	for (int j = vertexArray[n-1]; j < m; ++j)
	{
			cout<<"e "<<n<<" "<<neighbourArray[j]<<endl;
				/* code */
		
	}
}

int main(int argc, char const *argv[])
{
	/* code */
	string a, b;
	int n, m;
	
	cin>>a>>b>>n>>m;
	
	// cout<<a<<" "<<b<<" "<<n<<" "<<m<<endl;

	int h_vertexArray[n];
	int h_neighbourArray[m];
	int h_degreeCount[n];
	
	int *d_vertexArray = NULL;
    hipMalloc((void **)&d_vertexArray, n*sizeof(int));
    
    int *d_neighbourArray = NULL;
    hipMalloc((void **)&d_neighbourArray, m*sizeof(int));
    	
    int *d_degreeCount = NULL;
    hipMalloc((void **)&d_degreeCount, (n)*sizeof(int));
    hipMemset((void *)d_degreeCount, 0, (n)*sizeof(int));
    	
	for (int i = 0; i < n; ++i)
	{
		/* code */
		h_vertexArray[i]=m;
	}

//	int offset = 0;

//	int current = 0;
//	int mark = 1;

//	for (int i = 0; i < m; ++i)
//	{
//		/* code */
//		char c;
//		int start;
//		int end;

//		cin>>start>>end;

//		if (start!=mark){ 

//			if (start == mark+1 && h_vertexArray[mark-1]!=m){ 

//			}

//			else{

//				for (int j = mark; j<start; j++){ 
//					h_vertexArray[j-1]=offset;
//					// h_neighbourArray[offset]=0;
//					// offset++;
//				}
//			}
//			mark = start;

//		}

//		if (start==current){ 
//			h_neighbourArray[offset]=end;
//			offset++;
//		}

//		else { 
//			current = start;

//			h_vertexArray[current-1]=offset;

//			h_neighbourArray[offset]=end;
//			offset++;
//		}
//	}

	int NSlast = 0;
	int NSoffset = 0;
	int NSprev=0;
	
	
	for (int i=0; i<m; i++){
		int start, end;
		cin>>start>>end;
		
		for (int j=NSlast+1; j<start; j++){
			h_vertexArray[j-1]=NSoffset;
			
		}
		
		if (NSprev!=start){
			NSlast=start;
			h_vertexArray[start-1]=NSoffset;
			NSprev=start;
		}
		
		h_neighbourArray[NSoffset]=end;
		NSoffset++;
		
	}
	
	
	hipMemcpy(d_vertexArray, h_vertexArray, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_neighbourArray, h_neighbourArray, m*sizeof(int), hipMemcpyHostToDevice);
	
	int threadsPerBlock = 512;
	int blocksPerGrid = (n + threadsPerBlock -1)/threadsPerBlock;
	
	//cout<<threadsPerBlock<<" "<<blocksPerGrid<<endl;

	degreeCalc<<<blocksPerGrid, threadsPerBlock>>>(d_vertexArray, d_neighbourArray, d_degreeCount, n, m);
	
	hipMemcpy(h_degreeCount, d_degreeCount, n*sizeof(int), hipMemcpyDeviceToHost);

	for (int i=0; i<n; i++){
		cout<<h_degreeCount[i]<<endl;
	}
	
	thrust::device_ptr<int> d_ptr = thrust::device_pointer_cast(d_degreeCount);
  	int max = *(thrust::max_element(d_ptr, d_ptr + n));
	
//	int result = thrust::reduce(h_degreeCount, h_degreeCount + n,
//                           -1,
//                            thrust::maximum<int>());
                            
//        cout<<"Result: "<<result<<endl<<max;

	cout<<max;

	//edgesPrint(h_vertexArray, h_neighbourArray, n, m);

	//delete[] h_vertexArray;
	//delete[] h_neighbourArray;
	//delete[] h_degreeCount;
	
	hipFree(d_neighbourArray);
	hipFree(d_vertexArray);
	hipFree(d_degreeCount);
	
	hipDeviceReset();
	return 0;
}
