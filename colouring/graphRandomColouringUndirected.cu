#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>


#include <hiprand/hiprand_kernel.h>

#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>

using namespace std;

__global__ void setup_kernel (hiprandState * state, unsigned long seed )
{
    int i= blockDim.x * blockIdx.x + threadIdx.x;

    hiprand_init (seed, i, 0, &state[i]);
} 

__global__ void randomColouring (hiprandState* globalState, int *degreeCount, int n, int limit){

	int i= blockDim.x * blockIdx.x + threadIdx.x;
	
	hiprandState localState = globalState[i];
    	float RANDOM = hiprand_uniform( &localState );
    	globalState[i] = localState;
    	
    	RANDOM *= (limit - 1 + 0.999999);
    	RANDOM += 1;
	
	degreeCount[i] = (int) RANDOM;
}

__global__ void conflictDetection (int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m, int *detectConflict){

	int i= blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i>=n){
		return;
	}
	
	int myColour = degreeCount[i];
	
	int start = -1, stop = -1;
	
	start = vertexArray[i];
	
	if (i==n-1){	
		stop = 2*m;
	}
	
	else{
		stop = vertexArray[i+1];
	}
	
	for (int j=start; j<stop; j++){
		if (degreeCount[neighbourArray[j]-1] == myColour){

//			detectConflict[i]=1;
//			break;

			if (i < neighbourArray[j]-1){
				if (detectConflict[i]!=1){
					detectConflict[i]=1;
				}
			}
			else if (detectConflict[neighbourArray[j]-1]!=1){
				detectConflict[neighbourArray[j]-1]=1;
			}
			
			
			
			
			
			
//			if (detectConflict[i]!=1){
//				detectConflict[i]=1;
//			}
//			
//			if (detectConflict[neighbourArray[j]-1]!=1){
//				detectConflict[neighbourArray[j]-1]=1;
//			}
		}
	}
}


__global__ void degreeCalc (int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m){

	int i= blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i>=n){
		return;
	}
	
	
	int start = -1, stop = -1;
	int diff=0;
	
	start = vertexArray[i];
	
	if (i==n-1){	
		stop = 2*m;
	}
	
	else{
		stop = vertexArray[i+1];
	}

	diff = stop-start;
		
	degreeCount[i]=diff;
}

void edgesPrint (int vertexArray[], int neighbourArray[], int n, int m){ 

	for (int i=0; i<n-1; i++){
		for (int j = vertexArray[i]; j < vertexArray[i+1]; ++j){

				cout<<"e "<<i+1<<" "<<neighbourArray[j]<<endl;
				/* code */
		}
	}

	for (int j = vertexArray[n-1]; j < m; ++j)
	{
			cout<<"e "<<n<<" "<<neighbourArray[j]<<endl;
				/* code */
		
	}
}

int main(int argc, char const *argv[])
{
	/* code */
	string a, b;
	int n, m;
	
	cin>>a>>b>>n>>m;
	
	// cout<<a<<" "<<b<<" "<<n<<" "<<m<<endl;

	int *h_vertexArray = new int [n];
	int *h_neighbourArray = new int [2*m];
	int *h_degreeCount = new int [n];
	int *h_detectConflict = new int [n];

	
	int *d_vertexArray = NULL;
    	hipMalloc((void **)&d_vertexArray, n*sizeof(int));
    	
    	int *d_neighbourArray = NULL;
    	hipMalloc((void **)&d_neighbourArray, 2*m*sizeof(int));
    	
    	int *d_detectConflict = NULL;
    	hipMalloc((void **)&d_detectConflict, (n)*sizeof(int));
    	hipMemset((void *)d_detectConflict, 0, (n)*sizeof(int));
    	
    	int *d_degreeCount = NULL;
    	hipMalloc((void **)&d_degreeCount, (n)*sizeof(int));
    	hipMemset((void *)d_degreeCount, 0, (n)*sizeof(int));
    	
    	hiprandState* devStates;
    	hipMalloc ( &devStates, n*sizeof( hiprandState ) );
    	
	for (int i = 0; i < n; ++i)
	{
		/* code */
		h_vertexArray[i]=2*m;
	}

	int offset = 0;

	int current = 0;
	int mark = 1;

	for (int i = 0; i < 2*m; ++i)
	{
		/* code */
		int start;
		int end;

		cin>>start>>end;

		
		
//		Uncomment for SNAP graph datasets with nodes indexed from 0 to n-1
		
//		cin>>start>>end;
//		start++;
//		end++;

		if (start!=mark){ 

			if (start == mark+1 && h_vertexArray[mark-1]!=2*m){ 

			}

			else{

				for (int j = mark; j<start; j++){ 
					h_vertexArray[j-1]=offset;
					// h_neighbourArray[offset]=0;
					// offset++;
				}
			}
			mark = start;

		}

		if (start==current){ 
			h_neighbourArray[offset]=end;
			offset++;
		}

		else { 
			current = start;

			h_vertexArray[current-1]=offset;

			h_neighbourArray[offset]=end;
			offset++;
		}
	}
	
	
	hipMemcpy(d_vertexArray, h_vertexArray, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_neighbourArray, h_neighbourArray, 2*m*sizeof(int), hipMemcpyHostToDevice);
	
	int threadsPerBlock = 512;
	int blocksPerGrid = (n + threadsPerBlock -1)/threadsPerBlock;
	
	//cout<<threadsPerBlock<<" "<<blocksPerGrid<<endl;
	
	degreeCalc<<<blocksPerGrid, threadsPerBlock>>>(d_vertexArray, d_neighbourArray, d_degreeCount, n, m);
	
//	hipMemcpy(h_degreeCount, d_degreeCount, n*sizeof(int), hipMemcpyDeviceToHost);

//	for (int i=0; i<n; i++){
//		cout<<h_degreeCount[i]<<endl;
//	}
	
	thrust::device_ptr<int> d_ptr = thrust::device_pointer_cast(d_degreeCount);
  	int max = *(thrust::max_element(d_ptr, d_ptr + n));
	
//	int result = thrust::reduce(h_degreeCount, h_degreeCount + n,
//                           -1,
//                            thrust::maximum<int>());
                            
//        cout<<"Result: "<<result<<endl<<max;

	cout<<"Max = "<<max<<endl;


	setup_kernel <<<blocksPerGrid, threadsPerBlock>>> ( devStates, time(NULL) );
	
	// Except for Cliques and Odd Cycles, Brook's theorem states that only Max Degree colours are enough at most
	randomColouring<<<blocksPerGrid, threadsPerBlock>>>(devStates, d_degreeCount, n, max);

	hipMemcpy(h_degreeCount, d_degreeCount, n*sizeof(int), hipMemcpyDeviceToHost);

//	for (int i=0; i<n; i++){
//		cout<<h_degreeCount[i]<<endl;
//	}
	
	conflictDetection<<<blocksPerGrid, threadsPerBlock>>>(d_vertexArray, d_neighbourArray, d_degreeCount, n, m, d_detectConflict);
	
	thrust::device_ptr<int> d_detectConflict_ptr = thrust::device_pointer_cast(d_detectConflict);
  	int count1 = thrust::reduce(d_detectConflict_ptr, d_detectConflict_ptr + n);
  	
  	hipMemcpy(h_detectConflict, d_detectConflict, n*sizeof(int), hipMemcpyDeviceToHost);
	
//	for (int i=0; i<n; i++){
//		cout<<i+1<<": "<<h_detectConflict[i]<<endl;
//	}
	
	cout<<"Count: "<<count1<<endl;
	
	int countnew=0;
	
	
	for (int i=0; i<n-1; i++){
		
		if (h_detectConflict[i]==0){
			continue;
		}
		
		countnew++;
		
		bool usedColours[max+1];
		
		fill(usedColours, usedColours+max+1, false);
		
//		if (flag){
//			flag = false;
//			for (int j=0; j<n; j++){
//				cout<<usedColours[i]<<endl;
//			}
//		}
		
		int start = -1, stop = -1;
	
		start = h_vertexArray[i];
		
		stop = h_vertexArray[i+1];
		
//		cout<<"My id: "<<i<<endl;
//		
//		cout<<"My colour: "<<h_degreeCount[i]<<endl;
//		
//		cout<<"Neighbours"<<endl;
//		
		for (int j=start; j<stop; j++){
		
//			cout<<h_degreeCount[h_neighbourArray[j]-1]<<" ";
			usedColours[h_degreeCount[h_neighbourArray[j]-1]-1] = true;
		}
//		cout<<endl;
		
		for (int j=0; j<max+1; j++){
			if (usedColours[j]==false){
				h_degreeCount[i]=j+1;
//				cout<<"My new Colour: "<<j+1<<endl;
				break;
			}
		}
	}
	
	
	
	if (h_detectConflict[n-1]!=0){

		bool usedColours[max+1];
		
		countnew++;
		
		fill(usedColours, usedColours+max+1, false);
		
		int start = -1, stop = -1;
	
		start = h_vertexArray[n-1];
	
		stop = 2*m;
		
	
		for (int j=start; j<stop; j++){
			usedColours[h_degreeCount[h_neighbourArray[j]-1]-1] = true;
		}
		
		for (int j=0; j<max+1; j++){
			if (usedColours[j]==false){
				h_degreeCount[n-1]=j+1;
				break;
			}
		}
	}
	
//	cout<<"SHAMILASADJKAJSDKLJASHDKJASHLDKASJKD";
//	for (int i=0; i<n; i++){
//		cout<<h_degreeCount[i]<<endl;
//	}

//	for (int i=0; i<n-1; i++){
//		
//		int start = -1, stop = -1;
//	
//		start = h_vertexArray[i];
//		
//		stop = h_vertexArray[i+1];
//		
//		cout<<"My id: "<<i<<endl;
//		
//		cout<<"My colour: "<<h_degreeCount[i]<<endl;
//		
//		cout<<"Neighbours"<<endl;
//		
//		for (int j=start; j<stop; j++){
//			cout<<h_degreeCount[h_neighbourArray[j]-1]<<" ";
//		}
//	}
//	
//	
//	
//	if (h_detectConflict[n-1]!=0){

//		int start = -1, stop = -1;
//	
//		start = h_vertexArray[n-1];
//	
//		stop = m;
//		
//		cout<<"My id: "<<n-1<<endl;
//		
//		cout<<"My colour: "<<h_degreeCount[n-1]<<endl;
//		
//		cout<<"Neighbours"<<endl;
//		
//		for (int j=start; j<stop; j++){
//			cout<<h_degreeCount[h_neighbourArray[j]-1]<<" ";
//		}
//	}

	cout<<"Shamil"<<endl;
	
	hipMemset((void *)d_detectConflict, 0, (n)*sizeof(int));
	
	hipMemcpy(d_degreeCount, h_degreeCount, n*sizeof(int), hipMemcpyHostToDevice);



	conflictDetection<<<blocksPerGrid, threadsPerBlock>>>(d_vertexArray, d_neighbourArray, d_degreeCount, n, m, d_detectConflict);
	
	count1 = thrust::reduce(d_detectConflict_ptr, d_detectConflict_ptr + n);
  		
	cout<<"Count: "<<count1<<"    "<<countnew<<endl;
	
	

//	for (int i=0; i<n; i++){
//		if (h_degreeCount[i] == max+1){
//			cout<<"BUHAHAHAHAHAHHAHAHAHHAHA"<<endl;
//		}
//		
//		else if (h_degreeCount[i] == 1){
//			cout<<"LALLLALALALALALALALALLALA"<<endl;
//		}
//		cout<<h_degreeCount[i]<<endl;
//	}

//	for (int i=0; i<n; i++){
//		cout<<i+1<<": "<<h_detectConflict[i]<<endl;
//	}


	
	//edgesPrint(h_vertexArray, h_neighbourArray, n, m);

	//delete[] h_vertexArray;
	//delete[] h_neighbourArray;
	//delete[] h_degreeCount;
	
	delete[] h_vertexArray;
	delete[] h_neighbourArray;
	delete[] h_degreeCount;
	delete[] h_detectConflict;
	
	hipFree(d_neighbourArray);
	hipFree(d_vertexArray);
	hipFree(d_degreeCount);
	hipFree(d_detectConflict);
	
	hipDeviceReset();
	return 0;
}
