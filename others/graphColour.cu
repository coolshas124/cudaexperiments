
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cstdio>

using namespace std;

__global__ void matrixColour (float *a, float *b, int n){
	int j= blockDim.x * blockIdx.x + threadIdx.x;
	
	if(j<n){
		for (int i=0; i<n; i++){
			printf("Thread = %d ; i = %d ; %f\n",j+1,i+1,b[i]);
			if (a[j*n+i]==1){
				if (b[j]==b[i]){
					b[j]=-1;
					break;
				}
			}
		}
	}
	
//	int colour[10];
//	
//	memset(colour, 0, 10*sizeof(float));
	
//	if (j<n){
//		for (int i=0; i<n; i++){
//			//printf("Thread = %d ; i = %d ; %f\n",j+1,i+1,b[i]);
//			if (a[j*n+i]==1 && b[i]!=-1){
//				colour[(int)b[i]]=1;
//			}
//			
//			
////			if (i==j){
////				//atomicAdd(&b[i],1.0f);
////				b[i]+=1.0f;
////			}
//		}
//		
//		for (int i=0; i<n; i++){
//			if (colour[i]==0){
//				b[j]=i;
//				break;
//			}
//			
//			
//			
//		}
//		
//		
//		for (int i=0; i<n; i++){
//			printf("Thread = %d ; i = %d ; %f\n",j+1,i+1,b[i]);
//			
//			
//		}
//		
//	}
	
	
	

//	printf("I am thread no: %d from blocknumber: %d\n", threadIdx.x, blockIdx.x);
	
	//b[j] = j+1;
	
	
}

int main(){
	
	int noNodes = 10;
	int n = noNodes*noNodes;
	
	size_t size = n * sizeof(float);
	size_t sizeColouring = noNodes * sizeof(float);
	
	float *h_matrix = new float[n];
	float *h_colouring = new float[noNodes];
	float *h_colouring1 = new float[noNodes];
	
	
	
	
	int k=3;
	
	for (int i=0; i<noNodes; i++){
		h_colouring[i]=rand()%k;
		h_colouring1[i]=-1;
		for (int j=0; j<noNodes; j++){
			if (i==j){
				h_matrix[noNodes*i+j]=0;
			}
			
			else{
				h_matrix[noNodes*i+j]=rand()%2;
			}
		}
	}
	
	for (int i=0; i<noNodes; i++){
	
		for (int j=0; j<noNodes; j++){
		
			cout<<h_matrix[noNodes*i+j]<<" ";	
		}
		
		cout<<endl;
		
	}
	
	float *d_matrix = NULL;
    hipMalloc((void **)&d_matrix, size);
    
    float *d_colouring = NULL;
    hipMalloc((void **)&d_colouring, sizeColouring);
    
    hipMemcpy(d_matrix, h_matrix, size, hipMemcpyHostToDevice);
	hipMemcpy(d_colouring, h_colouring, sizeColouring, hipMemcpyHostToDevice);
	
	for (int i=0; i<noNodes; i++){
		cout<<"Back Home i = "<<i+1<<" ; "<<h_colouring[i]<<endl;
	}

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 512;
    int blocksPerGrid = (noNodes + threadsPerBlock -1)/threadsPerBlock;
    
    cout<<"Blocks "<<threadsPerBlock<<" "<<blocksPerGrid<<endl;
    
    matrixColour<<<blocksPerGrid, threadsPerBlock>>>(d_matrix, d_colouring, noNodes);
    
    hipMemcpy(h_colouring, d_colouring, sizeColouring, hipMemcpyDeviceToHost);

	for (int i=0; i<noNodes; i++){
		cout<<"Back Home i = "<<i+1<<" ; "<<h_colouring[i]<<endl;
	}
	
	bool colour[noNodes];
	
	memset(colour, 0, noNodes*sizeof(bool));
	
	for (int i=0; i<noNodes; i++){
		if (h_colouring[i]==-1){
			for (int j=0; j<noNodes; j++){
				if (h_matrix[i*noNodes+j]==1){
					if (h_colouring[j] >= 0)	
						colour[(int)h_colouring[j]]=true;
				}
			}
			
			for (int j=0; j<noNodes; j++){
				if (colour[j]==false){
					h_colouring[i]=j;
					break;
				}
			}
			memset(colour, 0, noNodes*sizeof(bool));
		}
	}
	
	for (int i=0; i<noNodes; i++){
		cout<<"Back Home i = "<<i+1<<" ; "<<h_colouring1[i]<<endl;
	}
	
	for (int i=0; i<noNodes; i++){
		if (h_colouring1[i]==-1){
			for (int j=0; j<noNodes; j++){
				if (h_matrix[i*noNodes+j]==1){
					if (h_colouring1[j] >= 0)	
						colour[(int)h_colouring1[j]]=true;
				}
			}
			
			for (int j=0; j<noNodes; j++){
				if (colour[j]==false){
					h_colouring1[i]=j;
					break;
				}
			}
			memset(colour, 0, noNodes*sizeof(bool));
		}
	}


	for (int i=0; i<noNodes; i++){
		cout<<"Back Home i = "<<i+1<<" ; "<<h_colouring[i]<<endl;
	}


	for (int i=0; i<noNodes; i++){
		cout<<"Back Home i = "<<i+1<<" ; "<<h_colouring1[i]<<endl;
	}
	
    hipFree(d_matrix);
	hipFree(d_colouring);
	
    free(h_colouring);
    free(h_matrix);

    hipDeviceReset();
			
	return 0;
}
